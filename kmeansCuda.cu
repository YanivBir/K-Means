#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <malloc.h>
#include <time.h>
#include <stdlib.h> 
#include <math.h>
#include "Point.h"
#include "mpi.h"

#define BIG_double (INFINITY)
#define MAX_CUDA_THREADS_BLOCK 512
#define MAX_POINTS_THREAD 100

hipError_t cuda_setCurrDistErr(hipError_t cudaStatus, char msg[], Point *points);
hipError_t cuda_findDiamErr(hipError_t cudaStatus, char msg[], Point* dev_clusterPoints, double *dev_distancesArr, double *distancesArr);
hipError_t cuda_calcPointsErr(hipError_t cudaStatus, char msg[], Point *dev_points, Point *dev_centers, double *dev_distance_output);

//calc distance between 2 points
__device__ double cuda_calcDistancePoints(Point *p1, Point *p2){
	double x, y, z;	
	x = p1->x - p2->x;
	y = p1->y - p2->y;
	z = p1->z - p2->z;
	x *= x;
	y *= y;
	z *= z;
	return sqrt(x + y + z);
}

//set the new posion of points at time
__device__ void cuda_setCurrentPosion(Point *p, double time){
	p->x = (p->x + (time*p->vX));
	p->y = (p->y + (time*p->vY));
	p->z = (p->z + (time*p->vZ));
	p->vX = 0;
	p->vY = 0;
	p->vZ = 0;
}

//updaete the new points posion
__global__ void setCurrentDistance(Point *points, int n, double time){
	int pointId = (blockIdx.x * MAX_CUDA_THREADS_BLOCK) + threadIdx.x;
	if (pointId < n) {
		cuda_setCurrentPosion(&points[pointId], time);
	}
}

//clusterCount is num of points in the cluster
__global__ void findMaxDistance(Point *dev_clusterPoints, int clusterCount, double *dev_distancesArr, int index) {
	int pointId = (blockIdx.x * blockDim.x) + threadIdx.x;
	int pointOffset;
	if (pointId < clusterCount)
	{
		pointOffset = pointId + index;
		for (int i = pointOffset +1; i < pointOffset+1+MAX_POINTS_THREAD; i++)
		{
			double curDistance = cuda_calcDistancePoints(&dev_clusterPoints[pointId], &dev_clusterPoints[i%clusterCount]);
			if (curDistance > dev_distancesArr[pointId])
				dev_distancesArr[pointId] = curDistance;
		}
	}
}

__global__ void pointsDistance(Point *dev_points, int n, Point *dev_centers, int k, double *dev_distance_output) {
	int pointId = (blockIdx.x * MAX_CUDA_THREADS_BLOCK) + threadIdx.x;
	if (pointId < n) {
		for (int j = 0; j < k; j++) { // for each cluster
			dev_distance_output[pointId*k + j] = cuda_calcDistancePoints(&dev_points[pointId], &dev_centers[j]);
		}
	}
}

//calc the current position
hipError_t cuda_setCurrentPosition(Point *points, int n, double time){
	Point *dev_points = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cuda_setCurrDistErr(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n", dev_points);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		cuda_setCurrDistErr(cudaStatus, "hipMalloc dev_points failed!\n", dev_points);
		return cudaStatus;
	}

	cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cuda_setCurrDistErr(cudaStatus, "hipMemcpyHostToDevice dev_points failed!\n", dev_points);
		return cudaStatus;
	}

	int cudaBlocks = n / MAX_CUDA_THREADS_BLOCK;
	if (n % MAX_CUDA_THREADS_BLOCK != 0)
		cudaBlocks++;
	setCurrentDistance << <cudaBlocks, MAX_CUDA_THREADS_BLOCK >> >(dev_points, n, time);

	cudaStatus = hipMemcpy(points, dev_points, n * sizeof(Point), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cuda_setCurrDistErr(cudaStatus, "hipMemcpyHostToDevice dev_points failed!\n", dev_points);
		return cudaStatus;
	}

	hipFree(dev_points);
	return cudaStatus;
}

//calc the points distance
hipError_t cuda_calcPointsDistance(Point *points, int n, Point *centers, int k, double *distance_output){
	Point *dev_points = 0;
	Point *dev_centers = 0;
	double *dev_distance_output = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cuda_calcPointsErr(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n", dev_points, dev_centers, dev_distance_output);
		return cudaStatus;
	}
	
	cudaStatus = hipMalloc((void**)&dev_points, n * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		cuda_calcPointsErr(cudaStatus, "hipMalloc dev_points failed!\n", dev_points, dev_centers, dev_distance_output);
		return cudaStatus;
	}
	
	cudaStatus = hipMemcpy(dev_points, points, n * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cuda_calcPointsErr(cudaStatus, "hipMemcpyHostToDevice dev_points failed!\n", dev_points, dev_centers, dev_distance_output);
		return cudaStatus;
	}
	
	cudaStatus = hipMalloc((void**)&dev_centers, k * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		cuda_calcPointsErr(cudaStatus, "hipMalloc dec_centers failed!\n", dev_points, dev_centers, dev_distance_output);
		return cudaStatus;
	}
	
	cudaStatus = hipMemcpy(dev_centers, centers, k * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cuda_calcPointsErr(cudaStatus, "hipMemcpyHostToDevice dec_centers failed!\n", dev_points, dev_centers, dev_distance_output);
		return cudaStatus;
	}
	
	cudaStatus = hipMalloc((void**)&dev_distance_output, n*k * sizeof(double));
	if (cudaStatus != hipSuccess) {
		cuda_calcPointsErr(cudaStatus, "hipMalloc dev_distance_output failed!\n", dev_points, dev_centers, dev_distance_output);
		return cudaStatus;
	}
	
	int cudaBlocks = n / MAX_CUDA_THREADS_BLOCK;
	if (n % MAX_CUDA_THREADS_BLOCK != 0)
		cudaBlocks++;

	pointsDistance << <cudaBlocks, MAX_CUDA_THREADS_BLOCK >> >(dev_points, n, dev_centers, k, dev_distance_output);
	cudaStatus = hipMemcpy(distance_output, dev_distance_output, n*k * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cuda_calcPointsErr(cudaStatus, "hipMemcpyHostToDevice dev_distance_output failed!\n", dev_points, dev_centers, dev_distance_output);
		return cudaStatus;
	}

	hipFree(dev_points);
	hipFree(dev_centers);
	hipFree(dev_distance_output);
	return cudaStatus;
}

//clusterCount is the num of the points that ralated to the cluster
hipError_t cuda_findDiameter(Point *clusterPoints, int clusterCount, double *diamter){
	Point *dev_clusterPoints = 0;
	double *dev_distancesArr = 0;
	double *distancesArr = 0;
	hipError_t cudaStatus;

	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		cuda_findDiamErr(cudaStatus, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}

	cudaStatus = hipMalloc((void**)&dev_clusterPoints, clusterCount * sizeof(Point));
	if (cudaStatus != hipSuccess) {
		cuda_findDiamErr(cudaStatus, "hipMalloc dev_clusterPoints failed!\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}
	cudaStatus = hipMemcpy(dev_clusterPoints, clusterPoints, clusterCount * sizeof(Point), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cuda_findDiamErr(cudaStatus, "hipMemcpyHostToDevice dev_clusterPoints failed!\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}
	cudaStatus = hipMalloc((void**)&dev_distancesArr, clusterCount * sizeof(double));
	if (cudaStatus != hipSuccess) {
		cuda_findDiamErr(cudaStatus, "hipMalloc dev_distancesArr failed!\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}

	distancesArr = (double *)malloc(sizeof(double)*clusterCount);
	if (distancesArr == NULL) {
		cuda_findDiamErr(cudaStatus, "distancesArr malloc  failed!\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}
	for (int index = 0; index < clusterCount; index++)
	{
		distancesArr[index] = 0;
	}

	cudaStatus = hipMemcpy(dev_distancesArr, distancesArr, clusterCount * sizeof(double), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		cuda_findDiamErr(cudaStatus, "hipMemcpyHostToDevice dev_distancesArr failed!\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}

	int cudaBlocks = clusterCount / MAX_CUDA_THREADS_BLOCK;
	if (clusterCount % MAX_CUDA_THREADS_BLOCK != 0)
		cudaBlocks++;

	for (int i = 0; i < clusterCount; i += MAX_POINTS_THREAD)
		findMaxDistance << <cudaBlocks, MAX_CUDA_THREADS_BLOCK >> >(dev_clusterPoints, clusterCount, dev_distancesArr, i);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		cuda_findDiamErr(cudaStatus, "findMaxDistance launch failed!\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}
	hipFree(dev_clusterPoints);

	// Waits until all threads done
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		cuda_findDiamErr(cudaStatus, "hipDeviceSynchronize failed!\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}

	//Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(distancesArr, dev_distancesArr, clusterCount * sizeof(double), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		cuda_findDiamErr(cudaStatus, "hipMemcpy  failed!\n", dev_clusterPoints, dev_distancesArr, distancesArr);
		return cudaStatus;
	}
	hipFree(dev_distancesArr);

	double maxDiamter = distancesArr[0];
	for (int i = 1; i < clusterCount; i++)
	{
		if (distancesArr[i] > maxDiamter)
			maxDiamter = distancesArr[i];
	}
	*diamter = maxDiamter;
	free(distancesArr);

	return cudaStatus;
}

hipError_t cuda_setCurrDistErr(hipError_t cudaStatus, char msg[], Point *points){
	hipFree(points);
	fprintf(stderr, msg);
	return cudaStatus;
}

hipError_t cuda_findDiamErr(hipError_t cudaStatus, char msg[], Point* dev_clusterPoints, double *dev_distancesArr, double *distancesArr){
	hipFree(dev_clusterPoints);
	hipFree(dev_distancesArr);
	free(distancesArr);
	fprintf(stderr, msg);
	return cudaStatus;
}

hipError_t cuda_calcPointsErr(hipError_t cudaStatus, char msg[], Point *dev_points, Point *dev_centers, double *dev_distance_output){
	hipFree(dev_points);
	hipFree(dev_centers);
	hipFree(dev_distance_output);
	fprintf(stderr, msg);
	return cudaStatus;
}

int finalizeCuda(){
	hipError_t cudaStatus;
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}
	return 0;
}
